#include "hip/hip_runtime.h"
// --------------------------------------------------------
// Multitask Network Cascade
// Written by Haozhi Qi
// Copyright (c) 2016, Haozhi Qi
// Licensed under The MIT License [see LICENSE for details]
// --------------------------------------------------------

#include "caffe/layers/mask_resize_layer.hpp"

namespace caffe {
  
template <typename Dtype>
__device__ Dtype bilinear_interpolate(const Dtype* bottom_data, const int input_height, const int input_width, Dtype inverse_y, Dtype inverse_x) {
  
  // deal with cases that inverse elements are out of feature map boundary
  if (inverse_y < -0.5 || inverse_y > input_height - 0.5 || inverse_x < -0.5 || inverse_x > input_width - 0.5) {
    return 0.0;
  }
  
  if (inverse_y <= 0) inverse_y = 0;
  if (inverse_x <= 0) inverse_x = 0;
  
  int h_low = (int) inverse_y;
  int w_low = (int) inverse_x;
  int h_high;
  int w_high;
  
  if (h_low >= input_height - 1) {
    h_high = h_low = input_height - 1;
    inverse_y = (Dtype) h_low;
  } else {
    h_high = h_low + 1;
  }
  
  if (w_low >= input_width - 1) {
    w_high = w_low = input_width - 1;
    inverse_x = (Dtype) w_low;
  } else {
    w_high = w_low + 1;
  }
  
  Dtype lh = inverse_y - h_low;
  Dtype lw = inverse_x - w_low;
  Dtype hh = 1 - lh, hw = 1 - lw;
  // do bilinear interpolation
  Dtype v1 = bottom_data[h_low * input_width + w_low];
  Dtype v2 = bottom_data[h_low * input_width + w_high];
  Dtype v3 = bottom_data[h_high * input_width + w_low];
  Dtype v4 = bottom_data[h_high * input_width + w_high];
  Dtype w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  Dtype val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}

template <typename Dtype>
__global__ void MaskResizeForward(const int nthreads, const Dtype* bottom_data, const int output_width, const int output_height, const int output_channels, const int input_width, const int input_height, const int input_channels, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // (n, c, h, w) is an element in output mask
    int w = index % output_width;
    int h = (index / output_width) % output_height;
    int c = (index / output_width / output_height) % output_channels;
    int n = index / output_width / output_height / output_channels;
    Dtype ratio_h = static_cast<Dtype>(input_height) / static_cast<Dtype>(output_height);
    Dtype ratio_w = static_cast<Dtype>(input_width) / static_cast<Dtype>(output_width);

 //   Dtype inverse_x = w * ratio_w;
 //  Dtype inverse_y = h * ratio_h;
    Dtype inverse_x = (w + 0.5) * ratio_w - 0.5;
    Dtype inverse_y = (h + 0.5) * ratio_h - 0.5;

    const Dtype* offset_bottom_data = bottom_data + (n * input_channels + c) * input_height * input_width; 
    top_data[index] = bilinear_interpolate(offset_bottom_data, input_height, input_width, inverse_y, inverse_x);
  }
} 
template <typename Dtype>
void MaskResizeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();
  MaskResizeForward<Dtype> <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
  (count, bottom_data, output_width_, output_height_, output_channels_, 
  input_width_, input_height_, input_channels_, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__device__ Dtype getGradientWeight(Dtype argmax_h, Dtype argmax_w, const int h, const int w, const int height, const int width){
  if (argmax_h < -0.5 || argmax_h >(height - 0.5) || argmax_w < -0.5 || argmax_w >(width - 0.5)){
      return 0;
    }
  
  if (argmax_h < 0) argmax_h = 0;
  if (argmax_w < 0) argmax_w = 0;
  
  int argmax_h_low = (int)argmax_h;
  int argmax_w_low = (int)argmax_w;
  int argmax_h_high;
  int argmax_w_high;
  if (argmax_h_low >= height - 1) {
    argmax_h_high = argmax_h_low = height - 1;
    argmax_h = (Dtype)argmax_h_low;
  }
  else
    argmax_h_high = argmax_h_low + 1;
  
  if (argmax_w_low >= width - 1) {
    argmax_w_high = argmax_w_low = width - 1;
    argmax_w = (Dtype)argmax_w_low;
  }
  else
    argmax_w_high = argmax_w_low + 1;
  
  Dtype weight = 0;
  if (h == argmax_h_low) {
    if (w == argmax_w_low) {
      weight = (h + 1 - argmax_h) * (w + 1 - argmax_w);
    }
    else if (w == argmax_w_high) {
      weight = (h + 1 - argmax_h) * (argmax_w + 1 - w);
    }
  }
  else if (h == argmax_h_high) {
    if (w == argmax_w_low) {
      weight = (argmax_h + 1 - h) * (w + 1 - argmax_w);
    }
    else if (w == argmax_w_high) {
      weight = (argmax_h + 1 - h) * (argmax_w + 1 - w);
    }
  }
  return weight;
}
template <typename Dtype>
__global__ void MaskResizeBackward(const int nthreads, const Dtype* top_diff,
          const int channels,const int height, const int width, const int pooled_height,
          const int pooled_width, Dtype* bottom_diff){
  CUDA_KERNEL_LOOP(index, nthreads){
    // (n,c,ph,pw) in top[0] feature map
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    Dtype weight_upper_left,weight_upper_right,weight_lower_left,weight_lower_right;

    Dtype index_h = (ph + 0.5) * height / pooled_height - 0.5;
    Dtype index_w = (pw + 0.5) * width / pooled_width - 0.5;

    weight_upper_left = getGradientWeight(index_h,index_w,(int)index_h,(int)index_w,height,width);
    weight_upper_right = getGradientWeight(index_h,index_w,(int)index_h,(int)index_w+1,height,width);
    weight_lower_left = getGradientWeight(index_h,index_w,(int)index_h+1,(int)index_w,height,width);
    weight_lower_right = getGradientWeight(index_h,index_w,(int)index_h+1,(int)index_w+1,height,width);

    const int offset_upper_left = ((n * channels + c) * height + (int)index_h) * width + (int)index_w;
    const int offset_upper_right = ((n * channels + c) * height + (int)index_h) * width + (int)index_w + 1;
    const int offset_lower_left = ((n * channels + c) * height + (int)index_h + 1) * width +(int)index_w;
    const int offset_lower_right = ((n * channels + c) * height + (int)index_h + 1) * width +(int)index_w + 1;

    bottom_diff[offset_upper_left] += weight_upper_left * top_diff[index];
    bottom_diff[offset_upper_right] += weight_upper_right * top_diff[index];
    bottom_diff[offset_lower_left] += weight_lower_left * top_diff[index];
    bottom_diff[offset_lower_right] += weight_lower_right * top_diff[index];
  }
}

template <typename Dtype>
void MaskResizeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, 
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom){
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    int count = top[0]->count();
    caffe_gpu_set(bottom[0]->count(), Dtype(0.), bottom_diff);
    if(propagate_down[0]){
      MaskResizeBackward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >
      (count, top_diff, input_channels_, input_height_, 
      input_width_, output_height_,output_width_, bottom_diff);
    }
    CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(MaskResizeLayer);

}