#include "hip/hip_runtime.h"
#include <cfloat>
#include "caffe/fast_rcnn_layers.hpp"
using std::max;
using std::min;
namespace caffe {
template <typename Dtype>
__device__ void bilinear_interpolate(const Dtype* bottom_data, const int height, const int width, Dtype h, Dtype w, Dtype & value) {
  // deal with cases that inverse elements are out of feature map boundary
  if (h < -0.5 || h > height - 0.5 || w < -0.5 || w > width - 0.5) {
    //empty
    return;
  }
  
  if (h <= 0) h = 0;
  if (w <= 0) w = 0;
  
  int h_low = (int) h;
  int w_low = (int) w;
  int h_high;
  int w_high;
  
  if (h_low >= height - 1) {
    h_high = h_low = height - 1;
    h = (Dtype) h_low;
  } else {
    h_high = h_low + 1;
  }
  
  if (w_low >= width - 1) {
    w_high = w_low = width - 1;
    w = (Dtype) w_low;
  } else {
    w_high = w_low + 1;
  }
  
  Dtype lh = h - h_low;
  Dtype lw = w - w_low;
  Dtype hh = 1 - lh, hw = 1 - lw;
  // do bilinear interpolation
  Dtype v1 = bottom_data[h_low * width + w_low];
  Dtype v2 = bottom_data[h_low * width + w_high];
  Dtype v3 = bottom_data[h_high * width + w_low];
  Dtype v4 = bottom_data[h_high * width + w_high];
  Dtype w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw; 
  value = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
}
template <typename Dtype>
__global__ void InterpolateForward(const int nthreads, const Dtype* bottom_data, const int channels, 
            const int height, const int width,const int pooled_height, const int pooled_width,
            Dtype* top_data){
  CUDA_KERNEL_LOOP(index, nthreads){
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    Dtype value = 0;

    Dtype index_h = (ph + 0.5) * height/pooled_height - 0.5;
    Dtype index_w = (pw + 0.5) * height/pooled_height - 0.5;

    bottom_data += (n * channels + c) * height * width;
    bilinear_interpolate(bottom_data,height,width,index_h,index_w,value);
    top_data[index] = value;
  }
}
template <typename Dtype>
void InterpolateLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
             const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();
  InterpolateForward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >
    (count, bottom_data, channels_, height_, width_, pooled_height_,pooled_width_, top_data);
  CUDA_POST_KERNEL_CHECK;
}
template<typename Dtype>
__device__ Dtype get_feature_gradient(Dtype argmax_h, Dtype argmax_w, const int h,
      const int w, const int height, const int width){
  if (argmax_h < -0.5 || argmax_h >(height - 0.5) || argmax_w < -0.5 || argmax_w >(width - 0.5)){
      //empty
      return 0;
  }
  if (argmax_h < 0) argmax_h = 0;
  if (argmax_w < 0) argmax_w = 0;
  
  int argmax_h_low = (int)argmax_h;
  int argmax_w_low = (int)argmax_w;
  int argmax_h_high;
  int argmax_w_high;
  if (argmax_h_low >= height - 1) {
    argmax_h_high = argmax_h_low = height - 1;
    argmax_h = (Dtype)argmax_h_low;
  }
  else
    argmax_h_high = argmax_h_low + 1;
  
  if (argmax_w_low >= width - 1) {
    argmax_w_high = argmax_w_low = width - 1;
    argmax_w = (Dtype)argmax_w_low;
  }
  else
    argmax_w_high = argmax_w_low + 1;
  
  Dtype weight = 0;
  if (h == argmax_h_low) {
    if (w == argmax_w_low) {
      weight = (h + 1 - argmax_h) * (w + 1 - argmax_w);
    }
    else if (w == argmax_w_high) {
      weight = (h + 1 - argmax_h) * (argmax_w + 1 - w);
    }
  }
  else if (h == argmax_h_high) {
    if (w == argmax_w_low) {
      weight = (argmax_h + 1 - h) * (w + 1 - argmax_w);
    }
    else if (w == argmax_w_high) {
      weight = (argmax_h + 1 - h) * (argmax_w + 1 - w);
    }
  }
  return weight;
}    
template <typename Dtype>
__global__ void InterpolateBackwardFeature(const int nthreads, const Dtype* top_diff,
          const int channels,const int height, const int width, const int pooled_height,
          const int pooled_width, Dtype* bottom_diff){
  CUDA_KERNEL_LOOP(index, nthreads){
    // (n,c,ph,pw) in top[0] feature map
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    Dtype weight_upper_left,weight_upper_right,weight_lower_left,weight_lower_right;

    Dtype index_h = (ph + 0.5) * height / pooled_height - 0.5;
    Dtype index_w = (pw + 0.5) * width / pooled_width - 0.5;

    weight_upper_left = get_feature_gradient(index_h,index_w,(int)index_h,(int)index_w,height,width);
    weight_upper_right = get_feature_gradient(index_h,index_w,(int)index_h,(int)index_w+1,height,width);
    weight_lower_left = get_feature_gradient(index_h,index_w,(int)index_h+1,(int)index_w,height,width);
    weight_lower_right = get_feature_gradient(index_h,index_w,(int)index_h+1,(int)index_w+1,height,width);

    const int offset_upper_left = ((n * channels + c) * height + (int)index_h) * width + (int)index_w;
    const int offset_upper_right = ((n * channels + c) * height + (int)index_h) * width + (int)index_w + 1;
    const int offset_lower_left = ((n * channels + c) * height + (int)index_h + 1) * width +(int)index_w;
    const int offset_lower_right = ((n * channels + c) * height + (int)index_h + 1) * width +(int)index_w + 1;

    bottom_diff[offset_upper_left] += weight_upper_left * top_diff[index];
    bottom_diff[offset_upper_right] += weight_upper_right * top_diff[index];
    bottom_diff[offset_lower_left] += weight_lower_left * top_diff[index];
    bottom_diff[offset_lower_right] += weight_lower_right * top_diff[index];
  }
}
template <typename Dtype>
void InterpolateLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
              const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom){

  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  int count = top[0]->count();
  caffe_gpu_set(bottom[0]->count(), Dtype(0.), bottom_diff);
  if(propagate_down[0]){
    InterpolateBackwardFeature<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >
    (count, top_diff, channels_, height_, width_, pooled_height_,pooled_width_, bottom_diff);
  }
  CUDA_POST_KERNEL_CHECK;
}
INSTANTIATE_LAYER_GPU_FUNCS(InterpolateLayer);
}