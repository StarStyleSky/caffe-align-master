#include "hip/hip_runtime.h"
#include "caffe/layers/butterworthlayer.hpp"
#include "caffe/util/math_functions.hpp"
#include <algorithm>
#include <utility>
namespace caffe{
	template<typename Dtype>
	__global__ void Butterworth_Forward(const int nthreads,const Dtype* bottom_data,const Dtype eps, const int channels,
		const int height,const int width,const Dtype exp,const Dtype cut_off,Dtype* top_data){
		CUDA_KERNEL_LOOP(index, nthreads) {

    		int n = index / width / height / channels;

    		Dtype maxval,minval;
    		int offset = channels* height * width;
    		const Dtype* batch_data = bottom_data + n * offset;

    		maxval = batch_data[0];
    		minval = batch_data[0];
    		for(int i = 1;i<offset;++i){
    			if (maxval < batch_data[i]){
    				maxval = batch_data[i];
    			}
    			if (minval > batch_data[i]){
    				minval = batch_data[i];
    			}
    		} 
    		top_data[index] = (bottom_data[index] - minval)/(maxval-minval + eps);
    		top_data[index] = Dtype(1.) - Dtype(1.)/(Dtype(1.)+pow(static_cast<double>(bottom_data[index]/cut_off),static_cast<double>(exp)));
    	}
	}
	template<typename Dtype>
	void ButterworthLayer<Dtype>:: Forward_gpu(const vector<Blob<Dtype>*>& bottom,
	const vector<Blob<Dtype>*>& top){
		const Dtype* bottom_data = bottom[0]->gpu_data();
		Dtype* top_data = top[0]->mutable_gpu_data();		

		const int  count = bottom[0]->count();
		Dtype exponent = Dtype(2.)*Dtype(orders_);

		Butterworth_Forward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,bottom_data,eps_,channels_,height_,width_,exponent,cut_off_,top_data);
		CUDA_POST_KERNEL_CHECK;
	}
	template<typename Dtype>
	void ButterworthLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom){
		 NOT_IMPLEMENTED; 		
	}
	INSTANTIATE_LAYER_GPU_FUNCS(ButterworthLayer);
}