#include "hip/hip_runtime.h"
#include <cfloat>
#include "caffe/layers/psroi_align_pooling_layer.hpp"
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
using std::max;
using std::min;
namespace caffe {
template <typename Dtype>
__device__ void bilinear_interpolate(const Dtype* bottom_data, const int height, const int width, Dtype h, Dtype w, Dtype & value) {
  // deal with cases that inverse elements are out of feature map boundary
  if (h < -0.5 || h > height - 0.5 || w < -0.5 || w > width - 0.5) {
    //empty
    return;
  }
  
  if (h <= 0) h = 0;
  if (w <= 0) w = 0;
  
  int h_low = (int) h;
  int w_low = (int) w;
  int h_high;
  int w_high;
  
  if (h_low >= height - 1) {
    h_high = h_low = height - 1;
    h = (Dtype) h_low;
  } else {
    h_high = h_low + 1;
  }
  
  if (w_low >= width - 1) {
    w_high = w_low = width - 1;
    w = (Dtype) w_low;
  } else {
    w_high = w_low + 1;
  }
  
  Dtype lh = h - h_low;
  Dtype lw = w - w_low;
  Dtype hh = 1 - lh, hw = 1 - lw;
  // do bilinear interpolation
  Dtype v1 = bottom_data[h_low * width + w_low];
  Dtype v2 = bottom_data[h_low * width + w_high];
  Dtype v3 = bottom_data[h_high * width + w_low];
  Dtype v4 = bottom_data[h_high * width + w_high];
  Dtype w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;
  
  value = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
} 
template <typename Dtype>
__global__ void PSROIAlignPoolingForward(
    const int nthreads,
    const Dtype* bottom_data,
    const Dtype spatial_scale,
    const int channels,
    const int height, const int width,
    const int pooled_height, const int pooled_width,
    const Dtype* bottom_rois,
    const int output_dim, // the channels of the output feature map
    const int group_size, // the size of the output feature map. pooled_height = group_size,pooled_width = group_size
    Dtype* top_data) {
    CUDA_KERNEL_LOOP(index, nthreads) {
      // The output is in order (n, ctop, ph, pw)
      int pw = index % pooled_width;
      int ph = (index / pooled_width) % pooled_height;
      int ctop = (index / pooled_width / pooled_height) % output_dim;
      int n = index / pooled_width / pooled_height / output_dim;

      // [start, end) interval for spatial sampling
      bottom_rois += n * 5;
      int roi_batch_ind = bottom_rois[0];

      Dtype roi_start_w = static_cast<Dtype>(bottom_rois[1]) * spatial_scale;
      Dtype roi_start_h = static_cast<Dtype>(bottom_rois[2]) * spatial_scale;
      Dtype roi_end_w = static_cast<Dtype>(bottom_rois[3]) * spatial_scale;
      Dtype roi_end_h = static_cast<Dtype>(bottom_rois[4]) * spatial_scale;

      // Force too small ROIs to be 1x1
      Dtype roi_width = max(roi_end_w - roi_start_w,(Dtype)1);  // avoid 0
      Dtype roi_height = max(roi_end_h - roi_start_h, (Dtype)1);

      // Compute w and h at bottom
      Dtype bin_size_h = roi_height / static_cast<Dtype>(pooled_height);
      Dtype bin_size_w = roi_width / static_cast<Dtype>(pooled_width);

      Dtype hstart = static_cast<Dtype>(ph) * bin_size_h + roi_start_h;
      Dtype wstart = static_cast<Dtype>(pw)* bin_size_w  + roi_start_w;
      Dtype hend = static_cast<Dtype>(ph + 1) * bin_size_h + roi_start_h;
      Dtype wend = static_cast<Dtype>(pw + 1) * bin_size_w + roi_start_w;

      // Add roi offsets and clip to input boundaries
      hstart = min(max(hstart, (Dtype)0), (Dtype)height);
      hend = min(max(hend, (Dtype)0), (Dtype)height);
      wstart = min(max(wstart, (Dtype)0), (Dtype)width);
      wend = min(max(wend, (Dtype)0), (Dtype)width);

      bool is_empty = (hend <= hstart) || (wend <= wstart);

      int c = (ctop * group_size + ph) * group_size + pw;

      const Dtype *batch_data = bottom_data + (roi_batch_ind * channels + c) * height * width;
      Dtype sum = 0,value;
      for (int i = 0;i < 2; ++i){
        for(int j = 0;j < 2; ++j){
          	value = 0;
            Dtype h = hstart + i * bin_size_h;
            Dtype w = wstart + j * bin_size_w;
            bilinear_interpolate(batch_data,height,width,h,w,value);
            sum += value;
        }
      }
      top_data[index] = is_empty ? 0:sum / static_cast<Dtype>(4);
    }
  }
  template <typename Dtype>
  void PSROIAlignPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* bottom_rois = bottom[1]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    int count = top[0]->count();
    caffe_gpu_set(count, Dtype(0), top_data);

    // NOLINT_NEXT_LINE(whitespace/operators)
    PSROIAlignPoolingForward<Dtype> << <CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS >> >(count, bottom_data, spatial_scale_,
      channels_, height_, width_, pooled_height_,pooled_width_, 
      bottom_rois, output_dim_, group_size_,top_data);
    CUDA_POST_KERNEL_CHECK;
  }
template <typename Dtype>
__device__ Dtype get_feature_gradient(Dtype argmax_h, Dtype argmax_w, 
    const int h, const int w, const int height, const int width){
  if (argmax_h < -0.5 || argmax_h >(height - 0.5) || argmax_w < -0.5 || argmax_w >(width - 0.5)){
      //empty
      return 0;
    }
  
  if (argmax_h < 0) argmax_h = 0;
  if (argmax_w < 0) argmax_w = 0;
  
  int argmax_h_low = (int)argmax_h;
  int argmax_w_low = (int)argmax_w;
  int argmax_h_high;
  int argmax_w_high;
  if (argmax_h_low >= height - 1) {
    argmax_h_high = argmax_h_low = height - 1;
    argmax_h = (Dtype)argmax_h_low;
  }
  else
    argmax_h_high = argmax_h_low + 1;
  
  if (argmax_w_low >= width - 1) {
    argmax_w_high = argmax_w_low = width - 1;
    argmax_w = (Dtype)argmax_w_low;
  }
  else
    argmax_w_high = argmax_w_low + 1;
  
  Dtype weight = 0;
  if (h == argmax_h_low) {
    if (w == argmax_w_low) {
      weight = (h + 1 - argmax_h) * (w + 1 - argmax_w);
    }
    else if (w == argmax_w_high) {
      weight = (h + 1 - argmax_h) * (argmax_w + 1 - w);
    }
  }
  else if (h == argmax_h_high) {
    if (w == argmax_w_low) {
      weight = (argmax_h + 1 - h) * (w + 1 - argmax_w);
    }
    else if (w == argmax_w_high) {
      weight = (argmax_h + 1 - h) * (argmax_w + 1 - w);
    }
  }
  return weight;
}
  
  template <typename Dtype>
  __global__ void PSROIAlignBackwardFeature(
    const int nthreads,
    const Dtype* top_diff,
    const Dtype spatial_scale,
    const int channels,const int height, 
    const int width,const int group_size,
    const int pooled_height, const int pooled_width,
    const int output_dim,
    Dtype* bottom_diff,
    const Dtype* bottom_rois) {
    CUDA_KERNEL_LOOP(index, nthreads) {
      // The output is in order (n, ctop, ph, pw)

      int pw = index % pooled_width;
      int ph = (index / pooled_width) % pooled_height;
      int ctop = (index / pooled_width / pooled_height) % output_dim;
      int n = index / pooled_width / pooled_height / output_dim;

      // [start, end) interval for spatial sampling
      bottom_rois += n * 5;
      int roi_batch_ind = bottom_rois[0];
      Dtype roi_start_w = static_cast<Dtype>(bottom_rois[1]) * spatial_scale;
      Dtype roi_start_h = static_cast<Dtype>(bottom_rois[2]) * spatial_scale;
      Dtype roi_end_w = static_cast<Dtype>(bottom_rois[3]) * spatial_scale;
      Dtype roi_end_h = static_cast<Dtype>(bottom_rois[4]) * spatial_scale;

      // Force too small ROIs to be 1x1
      Dtype roi_width = max(roi_end_w - roi_start_w, (Dtype)1);  // avoid 0
      Dtype roi_height = max(roi_end_h - roi_start_h, (Dtype)1);

      // Compute w and h at bottom
      Dtype bin_size_h = roi_height / static_cast<Dtype>(pooled_height);
      Dtype bin_size_w = roi_width / static_cast<Dtype>(pooled_width);

      Dtype hstart = static_cast<Dtype>(ph)* bin_size_h + roi_start_h;
      Dtype wstart = static_cast<Dtype>(pw)* bin_size_w + roi_start_w;
      Dtype hend = static_cast<Dtype>(ph + 1) * bin_size_h + roi_start_h;
      Dtype wend = static_cast<Dtype>(pw + 1) * bin_size_w + roi_start_w;
      
      // Add roi offsets and clip to input boundaries
      hstart = min(max(hstart, (Dtype)0), (Dtype)height);
      hend = min(max(hend, (Dtype)0), (Dtype)height);
      wstart = min(max(wstart, (Dtype)0), (Dtype)width);
      wend = min(max(wend, (Dtype)0), (Dtype)width);

      bool is_empty = (hend <= hstart) || (wend <= wstart);

      // Compute c at bottom
      int c = (ctop * group_size + ph) * group_size + pw;
      int offset = (roi_batch_ind * channels + c) * height * width;

      for(int i = 0;i < 2; ++i){
        for(int j = 0;j < 2; ++j){

            Dtype index_h = hstart + i * bin_size_h;
            Dtype index_w = wstart + j * bin_size_w;

            Dtype weight_upper_left,weight_upper_right,weight_lower_left,weight_lower_right;

            weight_upper_left = get_feature_gradient(index_h,index_w,(int)index_h,(int)index_w,height,width);
            weight_upper_right = get_feature_gradient(index_h,index_w,(int)index_h,(int)index_w+1,height,width);
            weight_lower_left = get_feature_gradient(index_h,index_w,(int)index_h+1,(int)index_w,height,width);
            weight_lower_right = get_feature_gradient(index_h,index_w,(int)index_h+1,(int)index_w+1,height,width);

            const int offset_upper_left =  offset + ((int)index_h) * width + (int)index_w;
            const int offset_upper_right = offset + ((int)index_h) * width + (int)index_w + 1;
            const int offset_lower_left = offset + ((int)index_h + 1) * width +(int)index_w;
            const int offset_lower_right = offset + ((int)index_h + 1) * width +(int)index_w + 1;

            bottom_diff[offset_upper_left] += (is_empty) ? 0.: 0.25 * weight_upper_left * top_diff[index];
            bottom_diff[offset_upper_right] += (is_empty) ? 0.: 0.25 * weight_upper_right * top_diff[index];
            bottom_diff[offset_lower_left] += (is_empty) ? 0.: 0.25 * weight_lower_left * top_diff[index];
            bottom_diff[offset_lower_right] += (is_empty) ? 0.: 0.25 * weight_lower_right * top_diff[index];   
        }
      }
    }
  }  

template <typename Dtype>
void PSROIAlignPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
              const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    if (!propagate_down[0]) {
        return;
    }
    const Dtype* bottom_rois = bottom[1]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int bottom_count = bottom[0]->count();

    caffe_gpu_set(bottom[1]->count(), Dtype(0), bottom[1]->mutable_gpu_diff());
    caffe_gpu_set(bottom_count, Dtype(0), bottom_diff);
    
    const int count = top[0]->count();
    
    // NOLINT_NEXT_LINE(whitespace/operators)

    PSROIAlignBackwardFeature<Dtype> << <CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS >> >(count, top_diff, spatial_scale_,
      channels_, height_, width_,group_size_,pooled_height_, 
      pooled_width_, output_dim_, bottom_diff , bottom_rois);
    CUDA_POST_KERNEL_CHECK;
  
}
  
INSTANTIATE_LAYER_GPU_FUNCS(PSROIAlignPoolingLayer);
  
}  // namespace caffe