#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>

#include "caffe/layers/channel_wise_scale.hpp"
#include "caffe/util/math_functions.hpp"
namespace caffe{
template <typename Dtype>
__global__ void ChannelWiseScaleForward(const int nthreads,const Dtype *bottom_data,
		const Dtype *scale_data,const int channels,const int height,const int width,
		Dtype *top_data) {
	CUDA_KERNEL_LOOP(index, nthreads){
		// (n, c, ph, pw) is an element in the pooled output
    	int c = (index / height / width) % channels;
    	int n = index / height / width / channels;

    	Dtype scale = *(scale_data + n * channels + c);  // get the scaler
    	top_data[index] = scale * bottom_data[index];
	}
}
template <typename Dtype>
void ChannelWiseScaleLayer<Dtype> :: Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top){
	const Dtype* bottom_data = bottom[0]->gpu_data();
	const Dtype* scale_factors = bottom[1]->gpu_data();
  	Dtype* top_data = top[0]->mutable_gpu_data();
  	int count = bottom[0]->count();
  	ChannelWiseScaleForward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >
  	(count,bottom_data,scale_factors,channels_,height_,width_,top_data);
  	CUDA_POST_KERNEL_CHECK;
}
template<typename Dtype>
__global__ void FeatureBackward(const int nthreads,const Dtype *top_diff,const Dtype *scale_data,
	const int channels,const int height,const int width,Dtype *bottom_data_diff){
	CUDA_KERNEL_LOOP(index, nthreads){
		int c = (index / height / width) % channels;
    	int n = index / height / width / channels;
    	Dtype scale = *(scale_data + n *channels +  c);
    	bottom_data_diff[index] = scale * top_diff[index];
	}
}
template<typename Dtype>
__global__ void ScaleBackward(const int nthreads,const Dtype *top_diff,const Dtype *bottom_data,
	const int channels,const int height,const int width,Dtype *bottom_scale_diff){
	CUDA_KERNEL_LOOP(index, nthreads){
		//(n,c,ph,pw) in top_diff and bottom_data feature map
    	int c = (index / height / width) % channels;
    	int n = index / height / width / channels;

    	int offset = n * channels + c;
    	bottom_scale_diff[offset] += top_diff[index]*bottom_data[index];
	}
}
template<typename Dtype>
void ChannelWiseScaleLayer<Dtype> :: Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom){

	const Dtype* bottom_data = bottom[0]->gpu_data();
	const Dtype* scale_factors = bottom[1]->gpu_data();
	const Dtype* top_diff = top[0]->gpu_diff();
	Dtype *bottom_data_diff = bottom[0]->mutable_gpu_diff();
	Dtype *bottom_scale_diff = bottom[1]->mutable_gpu_diff();

	caffe_gpu_set(bottom[0]->count(),Dtype(0),bottom_data_diff);
	caffe_gpu_set(bottom[1]->count(),Dtype(0),bottom_scale_diff);
	
	int count = bottom[0]->count();
	if(propagate_down[0]){

		FeatureBackward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >
		(count,top_diff,scale_factors,channels_,height_,width_,bottom_data_diff);
	}
	if(propagate_down[1]){

		ScaleBackward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >
		(count,top_diff,bottom_data,channels_,height_,width_,bottom_scale_diff);
	}
	CUDA_POST_KERNEL_CHECK;
}
INSTANTIATE_LAYER_GPU_FUNCS(ChannelWiseScaleLayer);
}